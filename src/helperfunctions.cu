
#include <hip/hip_runtime.h>
/*******************************************************************************************
 * Filename: helperfunctions.cu
 * Author: Drew Hans (github.com/drewhans555)
 * Description: This file contains helper functions - simple array operations, dealing with
 *              critical errors, printing insults, etc.
 *******************************************************************************************
 */

/*
 * initArrayToRandomDoubles
 * @params: a - a pointer to an array of double values
 * @params: n - the size of array a
 */
void initArrayToRandomDoubles(double* a, int n) {
    // generate random doubles in range [0, 1)
    for (int i = 0; i < n; i++) {
        a[i] = ((double) rand()) / ((double) RAND_MAX);
    }
} //end initArrayToRandomDoubles method

/*
 * initArrayToZeros
 * @params: a - a pointer to an array of double values
 * @params: n - the size of array a
 */
void initArrayToZeros(double* a, int n) {
    // set all neuron values to zero
    for (int i = 0; i < n; i++) {
        a[i] = 0;
    }
} //end initArrayToZeros method

/*
 * printarray - prints out array values to terminal
 * @params: name - a pointer to a char string
 * @params: a - a pointer to an array of double values
 * @params: n - the size of array a
 */
void printarray(const char* name, double* a, int n) {
    for (int i = 0; i < n; i++) {
        printf("%s[%d]=%lf\n", name, i, a[i]);
    }
    printf("\n");
} //end printarray method

/*
 * printFarewellMSG - prints out one final insult before we crash
 */
void printFarewellMSG() {
    printf("Sorry, I did everything I could but it looks like I'm crashing...\n...\n...your computer sucks, good-bye.\n");
} //end printFarewellMSG method

/*
 * onFileOpenError - SOS, we're going down
 * @params: path - file that failed to open
 */
void onFileOpenError(const char* path) {
    printf("ERROR: Failed to open %s!\n", path);
    printFarewellMSG();
    exit(1);
} //end onFileOpenError method

/*
 * onFileReadError - SOS, we're going down
 * @params: path - file that failed to read
 */
void onFileReadError(const char* path) {
    printf("ERROR: Failed to read value from file %s!\n", path);
    printFarewellMSG();
    exit(1);
} //end onFileReadError method

/*
 * onInvalidInput - prints out insults when the user screws up (silly humans)
 * @params: myPatience - the current state of my patience, represented as an int
 */
void onInvalidInput(int myPatience) {
    if (myPatience == 2) {
        printf("Looks like you entered an illegal value... you're testing my patience, try again!\n");
    } else if (myPatience == 1) {
        printf("That's the second time you've entered an illegal value... do you think this is funny? Try again!\n");
    } else if (myPatience == 0) {
        printf("Sigh... you just can't do anything right, can you?\n");
    } else {
        printf("Look dude, I've got all day. If you wanna keep wasting your time then that's fine by me. You know what you're supposed to do.\n");
    }
} //end onInvalidInput method

/*
 * onMallocError - SOS, we're going down
 * @params: size - the size of the memory that we couldn't allocate
 */
void onMallocError(int size) {
    printf("ERROR: Failed to malloc %d of memory!\n", size);
    printFarewellMSG();
    exit(1);
} //end onMallocError method